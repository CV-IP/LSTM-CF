#include <vector>

#include "caffe/layers/last_row_layer.hpp"

namespace caffe {

template <typename Dtype>
void LastRowLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
	Dtype* top_data = top[0]->mutable_gpu_data();

	int num = bottom[0]->shape(0);
	int num1 = bottom[0]->shape(1);
	int channels = bottom[0]->shape(2);

	bottom_data += bottom[0]->offset(num - 1);
	caffe_copy(channels * num1, bottom_data, top_data);
}

template <typename Dtype>
void LastRowLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    const Dtype* top_diff = top[0]->gpu_diff();
	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

	int num = bottom[0]->shape(0);
	int num1 = bottom[0]->shape(1);
	int channels = bottom[0]->shape(2);

	bottom_diff += bottom[0]->offset(num - 1);
	caffe_copy(channels * num1, top_diff, bottom_diff);	

}

INSTANTIATE_LAYER_GPU_FUNCS(LastRowLayer);

}